#include "hip/hip_runtime.h"
// Copyright (c) 2020, 2021, NECSTLab, Politecnico di Milano. All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NECSTLab nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//  * Neither the name of Politecnico di Milano nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <sstream>
#include "personalized_pagerank.cuh"

namespace chrono = std::chrono;
using clock_type = chrono::high_resolution_clock;

//////////////////////////////
//////////////////////////////

// Write GPU kernel here!
int *x_gpu;
int *y_gpu;
float *val_gpu;
int *V_gpu;
int *E_gpu;
float *pr_gpu;
float *pr_tmp;
int *dangling_gpu;

int *y_csr;
int *x_ptr;
float *val_csr;

float *val_float;
float *pr_float;

//#define log_warp_size 5;
//#define warp_size 32;
void spmv_sim(
                    const int *x, //x_ptr
                    const int *y, 
                    const float *val, 
                    const float *vec, 
                    float *result,
                    int V,
                    int pers,
                    float alpha,
                    float beta,
                    int id) {
	float res = 0;
    if (id < V) {
        
        int els = x[id+1] - x[id];
        for(int i = 0; i < els; i++) {
            res += val[x[id] + i] * vec[y[x[id] + i]];
            //if (vec[y[x[id] - els +i]] != 0)
                //std::cout << "vec[y[x[id] - els +i]] = "<< vec[y[x[id] - els +i]] << std::endl;
            if (i == els -1){
                res = (res * alpha) + beta;
                //check personalization vector
                if (id == pers) {
                    res = res + (1 - alpha);
                }
            }
        }
        //syncthreads before write?
        
        if (res == 1.0/V){
            std::cout << "no change for pr_" << id << std::endl;
        }

        std::cout << "new pr of v_" << id << " : " << res << std::endl;
        result[id] = res;

    }
}

__global__ void spmv(
                    const int *x, //x_ptr
                    const int *y, 
                    const float *val, 
                    const float *vec, 
                    float *result,
                    int V,
                    int pers,
                    float alpha,
                    float beta) {
	int id = threadIdx.x+blockIdx.x*blockDim.x;
	float res = 0;
    if (id < V - 1) {
        int els = x[id] - (id == 0 ? 0 : x[id-1]);

        for(int i = 0; i < els; i++) {
            res += val[x[id] + i] * vec[y[x[id]+i]];

            if (x[i] != x[i + 1])
                res = (res * alpha) + beta;
            //check personalization vector
            if (x[id] == pers)
                res += 1 - alpha;
        }
        //syncthreads before write?
        result[id] = res;
    }
}

__global__ void dangle_factor(
                            const int *x, //x_ptr
                            const int *y, 
                            const float *pr,
                            int V,
                            float result) {

    extern __shared__ float buffer[];
	//assert(sizeof(buffer) >= ((V >> log_warp_size) + 1));
    int id = threadIdx.x+blockIdx.x*blockDim.x;
    int partial = 0;
    int warp_size = 32;
    int log_warp_size = 5;

    if (id < V - 1) {
        int els = x[id] - (id == 0 ? 0 : x[id-1]);
        if (els == 0) {
            partial = pr[y[x[id]]];
        }
        for (int offset = warp_size / 2; offset > 0; offset /= 2) {
            partial += __shfl_down_sync(0xffffffff, partial, offset);
        }
        
        if ((id % warp_size) == 0) {
            buffer[(id >> log_warp_size)] = partial;
        }

        els = (V << log_warp_size) + (V % warp_size);

        while (els > 1) {
            //if (id > els)
            //    return;
            partial = buffer[id];

            for (int offset = warp_size / 2; offset > 0; offset /= 2) {
                partial += __shfl_down_sync(0xffffffff, partial, offset);
            }

            if (id % (warp_size) == 0)
                buffer[id >> log_warp_size] = partial;
        
            els = (els << log_warp_size) + (els % warp_size);
        }
        result = buffer[0];
    }
}

//////////////////////////////
//////////////////////////////

// CPU Utility functions;

inline float dot_product_cpu_float(const int *a, const float *b, const int N) {
    float result = 0;
    for (int i = 0; i < N; i++) {
        result += a[i] * b[i];
    }
    return result;
}

inline void axpb_personalized_cpu_float(
    float alpha, float *x, float beta,
    const int personalization_vertex, float *result, const int N) {
    float one_minus_alpha = 1 - alpha;
    for (int i = 0; i < N; i++) {
        result[i] = alpha * x[i] + beta + ((personalization_vertex == i) ? one_minus_alpha : 0.0);
    }
}

inline float euclidean_distance_cpu_float(const float *x, const float *y, const int N) {
    double result = 0;
    for (int i = 0; i < N; i++) {
        float tmp = x[i] - y[i];
        result += tmp * tmp;
    }
    return std::sqrt(result);
}

bool sort_csr(const std::tuple<int, int, float>& a, 
              const std::tuple<int, int, float>& b)
{
    if (std::get<1>(a) != std::get<1>(b))
        return (std::get<1>(a) < std::get<1>(b));
    else
        return std::get<0>(a) < std::get<0>(b);
}

// Read the input graph and initialize it;
void PersonalizedPageRank::initialize_graph() {
    // Read the graph from an MTX file;
    int num_rows = 0;
    int num_columns = 0;
    read_mtx(graph_file_path.c_str(), &x, &y, &val,
        &num_rows, &num_columns, &E, // Store the number of vertices (row and columns must be the same value), and edges;
        true,                        // If true, read edges TRANSPOSED, i.e. edge (2, 3) is loaded as (3, 2). We set this true as it simplifies the PPR computation;
        false,                       // If true, read the third column of the matrix file. If false, set all values to 1 (this is what you want when reading a graph topology);
        debug,                 
        false,                       // MTX files use indices starting from 1. If for whatever reason your MTX files uses indices that start from 0, set zero_indexed_file=true;
        true                         // If true, sort the edges in (x, y) order. If you have a sorted MTX file, turn this to false to make loading faster;
    );
    if (num_rows != num_columns) {
        if (debug) std::cout << "error, the matrix is not squared, rows=" << num_rows << ", columns=" << num_columns << std::endl;
        exit(-1);
    } else {
        V = num_rows;
    }
    if (debug) std::cout << "loaded graph, |V|=" << V << ", |E|=" << E << std::endl;

    // Compute the dangling vector. A vertex is not dangling if it has at least 1 outgoing edge;
    dangling.resize(V);
    std::fill(dangling.begin(), dangling.end(), 1);  // Initially assume all vertices to be dangling;
    for (int i = 0; i < E; i++) {
        // Ignore self-loops, a vertex is still dangling if it has only self-loops;
        if (x[i] != y[i]) dangling[y[i]] = 0;
    }
    // Initialize the CPU PageRank vector;
    pr.resize(V);
    pr_golden.resize(V);
    // Initialize the value vector of the graph (1 / outdegree of each vertex).
    // Count how many edges start in each vertex (here, the source vertex is y as the matrix is transposed);
    int *outdegree = (int *) calloc(V, sizeof(int));
    for (int i = 0; i < E; i++) {
        outdegree[y[i]]++;
    }
    // Divide each edge value by the outdegree of the source vertex;
    for (int i = 0; i < E; i++) {
        val[i] = 1.0 / outdegree[y[i]];  
    }
    free(outdegree);
}

//////////////////////////////
//////////////////////////////

// Allocate data on the CPU and GPU;
void PersonalizedPageRank::alloc() {
    // Load the input graph and preprocess it;
    initialize_graph();

    // Allocate any GPU data here;
    // TODO!

    hipMalloc(&x_gpu, sizeof(int)*(V+1));
    hipMalloc(&y_gpu, sizeof(int)*E);
    hipMalloc(&val_gpu, sizeof(float)*E);
    hipMalloc(&dangling_gpu, sizeof(int)*dangling.size());
    hipMalloc(&pr_gpu, sizeof(float) * V);
    hipMalloc(&pr_tmp, sizeof(float) * V);
}

// Initialize data;
void PersonalizedPageRank::init() {
    // Do any additional CPU or GPU setup here;
    // TODO!
    
    //convert to CSR
    x_ptr = (int *) malloc(sizeof(int)*(V+1));
    memset(x_ptr, 0, sizeof(int) * V);

    for(int i = 0; i < E; i++) {
        x_ptr[x[i]+1]++;
    }

    for(int i = 0; i < V; i++) {
        x_ptr[i+1] += x_ptr[i];
    }

    /*
    int els = 41;
    std::cout << "x_csr: ";
    for(int i = 0; i < els; i++) {
        std::cout << x_csr[i] << ", ";
    }
    std::cout << std::endl << "y_ptr: ";
    for(int i = 0; i < els; i++) {
        if (i < V)
            std::cout << y_ptr[i] << ", ";
    }
    std::cout << std::endl << "val: ";
    for(int i = 0; i < els; i++) {
        std::cout << val_csr[i] << ", ";
    }
    */
}


// Reset the state of the computation after every iteration.
// Reset the result, and transfer data to the GPU if necessary;
void PersonalizedPageRank::reset() {
    // Reset the PageRank vector (uniform initialization, 1 / V for each vertex);
    std::fill(pr.begin(), pr.end(), 1.0 / V); 
    // Generate a new personalization vertex for this iteration;
    personalization_vertex = rand() % V; 
    if (debug) std::cout << "personalization vertex=" << personalization_vertex << std::endl;

    val_float = (float *) malloc(sizeof(float)*E);
    std::transform(val.begin(), val.end(), val_float, [](double d) -> float {return float(d);});

    float *pr_float = (float *) malloc(sizeof(float)*V);

    float smallest = std::max(float(1.0/V) , float(1.0e-30));
    if (debug)
        std::cout << "initializing pr to: " << smallest << std::endl;
    for (int i = 0; i < V; i++) {
        pr_float[i] = smallest;
    }
    // Do any GPU reset here, and also transfer data to the GPU;
    // TODO!
    hipMemcpy(x_gpu, x_ptr, sizeof(int) * (V+1), hipMemcpyHostToDevice);
    hipMemcpy(y_gpu, y.data(), sizeof(int) * E, hipMemcpyHostToDevice);
    hipMemcpy(val_gpu, val_csr, sizeof(float) * E, hipMemcpyHostToDevice);
    hipMemcpy(dangling_gpu, dangling.data(), sizeof(int)*dangling.size(), hipMemcpyHostToDevice);
    hipMemcpy(pr_gpu, pr_float, sizeof(float) * V, hipMemcpyHostToDevice);


    // Print out the vector
    std::cout << "E = " << E << " V = " << V << std::endl;

    if (debug) std::cout << "reset successful" << std::endl;
}

void PersonalizedPageRank::execute(int iter) {
    if (debug) std::cout << "starting execution" << std::endl;
    // Do the GPU computation here, and also transfer results to the CPU;
    //TODO! (and save the GPU PPR values into the "pr" array)
    int block_size = 128;
    int n_blocks = max(1, (int) std::ceil(V / block_size));
    //int sharedmemSize = sizeof(float) * ((V >> 5) + 1);
    
    float alpha_f = (float) alpha;
    double* pr_cpu = (double*) malloc(sizeof(double)*V);
    float* pr_tmp_gpu = (float*) malloc(sizeof(float)*V);
    float smallest = std::max(float(1.0/V) , float(1.0e-30));
    for (int i = 0; i < V; i++) {
        pr_cpu[i] = smallest;
    }

    double* pr_tmp_cpu = (double*) malloc(sizeof(double)*V);

    //spvm_sim////////////////////////////////
    float* pr_tmp_sim = (float*) malloc(sizeof(float)*V);
    float* pr_sim = (float*) malloc(sizeof(float)*V);
    for (int i = 0; i < V+1; i++) {
        pr_sim[i] = smallest;
    }
    //////////////////////////////////////////

    bool converged = false;
    while (iter < max_iterations) {    

        
        memset(pr_tmp_cpu, 0, sizeof(double) * V);
        
        spmv_coo_cpu(x.data(), y.data(), val.data(), pr_cpu, pr_tmp_cpu, E);
        double dangling_factor = dot_product_cpu(dangling.data(), pr_cpu, V);
        std::cout << "dang_cpu: "<< dangling_factor << std::endl;
        axpb_personalized_cpu(alpha, pr_tmp_cpu, alpha * dangling_factor / V, personalization_vertex, pr_tmp_cpu, V);
        
        memcpy(pr_cpu, pr_tmp_cpu, sizeof(double) * V);
        
        //if (debug) std::cout << "launching gpu kernel" << std::endl;
        hipMemset(pr_tmp, 0, sizeof(float) * V);
        
        float dang_gpu = 0;
        /*
        dangle_factor<<<n_blocks, block_size, sharedmemSize>>>(x_gpu, y_gpu, pr_gpu, V, dang_gpu);
        std::cout << "dang_gpu: "<< dang_gpu << std::endl;
        printf("error after dang : %s\n", hipGetErrorName(hipGetLastError()));
        printf("%s\n", hipGetErrorString(hipGetLastError()));
        */
        dang_gpu = (float) dangling_factor;
        /*
        spmv<<<n_blocks, block_size>>>(x_gpu, y_gpu, val_gpu, pr_gpu, pr_tmp, V, personalization_vertex,
                                    (float) alpha, (float) alpha * dang_gpu / V);
        printf("error after spmv : %s\n", hipGetErrorName(hipGetLastError()));
        printf("%s\n", hipGetErrorString(hipGetLastError()));
        
        //if (debug) std::cout << "exiting gpu kernel" << std::endl;
        
        pr_tmp_gpu = (float *) malloc(sizeof(float)*V);
        hipMemcpy(pr_tmp_gpu, pr_tmp, sizeof(float) * V, hipMemcpyDeviceToHost);
        */
        std::cout << "start simulation"<< std::endl;
        memset(pr_tmp_sim, 0, sizeof(float) * V);
        int i = 0;
        for(i = 0; i < V; i++) {
            //std::cout << "pr of v_" << i << " = " << pr_sim[i] << std::endl;
            spmv_sim(x_ptr, y.data(), val_float, pr_sim, pr_tmp_sim, V, personalization_vertex,
                                    (float) alpha, (float) alpha * dang_gpu / V, i);
        }       
        memcpy(pr_sim, pr_tmp_sim, sizeof(float) * V);
/*
        for (int i = 0; i < 20; i++) {
            if (iter < 2) {
            std::cout << "expected = " << pr_tmp_cpu[i] << std::endl;
            std::cout << "found    = " << pr_tmp_sim[i] << std::endl;
            }
        }*/
        

        hipMemcpy(pr_gpu, pr_tmp, sizeof(float) * V, hipMemcpyDeviceToDevice);
        
        iter++;
        if (debug) std::cout << "end iter: " << iter << std::endl;
    }
    memcpy(pr.data(), pr_sim, sizeof(double) * V);
    //hipMemcpy(pr.data(), pr_gpu, sizeof(float) * V, hipMemcpyDeviceToHost);
    free(pr_cpu);
}

void PersonalizedPageRank::cpu_validation(int iter) {

    // Reset the CPU PageRank vector (uniform initialization, 1 / V for each vertex);
    std::fill(pr_golden.begin(), pr_golden.end(), 1.0 / V);

    // Do Personalized PageRank on CPU;
    auto start_tmp = clock_type::now();
    personalized_pagerank_cpu(x.data(), y.data(), val.data(), V, E, pr_golden.data(), dangling.data(), personalization_vertex, alpha, 1e-6, 100);
    auto end_tmp = clock_type::now();
    auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
    std::cout << "exec time CPU=" << double(exec_time) / 1000 << " ms" << std::endl;

    // Obtain the vertices with highest PPR value;
    std::vector<std::pair<int, double>> sorted_pr_tuples = sort_pr(pr.data(), V);
    std::vector<std::pair<int, double>> sorted_pr_golden_tuples = sort_pr(pr_golden.data(), V);

    // Check how many of the correct top-20 PPR vertices are retrieved by the GPU;
    std::unordered_set<int> top_pr_indices;
    std::unordered_set<int> top_pr_golden_indices;
    int old_precision = std::cout.precision();
    std::cout.precision(4);
    int topk = std::min(V, topk_vertices);
    for (int i = 0; i < topk; i++) {
        int pr_id_gpu = sorted_pr_tuples[i].first;
        int pr_id_cpu = sorted_pr_golden_tuples[i].first;
        top_pr_indices.insert(pr_id_gpu);
        top_pr_golden_indices.insert(pr_id_cpu);
        if (debug) {
            double pr_val_gpu = sorted_pr_tuples[i].second;
            double pr_val_cpu = sorted_pr_golden_tuples[i].second;
            if (pr_id_gpu != pr_id_cpu) {
                std::cout << "* error in rank! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            } else if (std::abs(sorted_pr_tuples[i].second - sorted_pr_golden_tuples[i].second) > 1e-6) {
                std::cout << "* error in value! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            }
        }
    }
    std::cout.precision(old_precision);
    // Set intersection to find correctly retrieved vertices;
    std::vector<int> correctly_retrieved_vertices;
    set_intersection(top_pr_indices.begin(), top_pr_indices.end(), top_pr_golden_indices.begin(), top_pr_golden_indices.end(), std::back_inserter(correctly_retrieved_vertices));
    precision = double(correctly_retrieved_vertices.size()) / topk;
    if (debug) std::cout << "correctly retrived top-" << topk << " vertices=" << correctly_retrieved_vertices.size() << " (" << 100 * precision << "%)" << std::endl;
}

std::string PersonalizedPageRank::print_result(bool short_form) {
    if (short_form) {
        return std::to_string(precision);
    } else {
        // Print the first few PageRank values (not sorted);
        std::ostringstream out;
        out.precision(3);
        out << "[";
        for (int i = 0; i < std::min(20, V); i++) {
            out << pr[i] << ", ";
        }
        out << "...]";
        return out.str();
    }
}

void PersonalizedPageRank::clean() {
    // Delete any GPU data or additional CPU data;
    // TODO!
}
